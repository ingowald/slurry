#include "hip/hip_runtime.h"
// Copyright 2025 Ingo Wald
// SPDX-License-Identifier: Apache-2.0

#include "compositing.h"
#include "miniApp.h"
#include <hip/hip_runtime.h>

/* has to match the name used in the embed_ptx cmake macro used in CMakeFile */
extern "C" const char devCode_ptx[];

namespace miniApp {

  typedef compositing::Context<Fragment,FinalCompositingResult> CompositingContext;
  vec2i fbSize { 800, 600 };
  struct {
    vec3f from { 0, 0, -1 };
    vec3f at   { 0, 0, 0 };
    vec3f up   { 0, 1, 0 };
    float fovy = 20.f;
  } camera;

  void setCamera(PerLaunchData &launchData)
  {
    /* vvvv all stolen from pete shirley's RTOW */
    const float vfov = camera.fovy;
    const vec3f vup = camera.up;
    const float aspect = fbSize.x / float(fbSize.y);
    const float theta = vfov * ((float)M_PI) / 180.0f;
    const float half_height = tanf(theta / 2.0f);
    const float half_width = aspect * half_height;
    const float focusDist = 10.f;
    const vec3f origin = camera.from;
    const vec3f w = normalize(camera.from - camera.at);
    const vec3f u = normalize(cross(vup, w));
    const vec3f v = cross(w, u);
    const vec3f lower_left_corner
      = origin - half_width * focusDist*u - half_height * focusDist*v - focusDist * w;
    const vec3f horizontal = 2.0f*half_width *focusDist*u;
    const vec3f vertical   = 2.0f*half_height*focusDist*v;
    /* ^^^^ all stolen from pete shirley's RTOW */

    launchData.camera.org = origin;
    launchData.camera.dir_00 = lower_left_corner;
    launchData.camera.dir_dx = horizontal / fbSize.x;
    launchData.camera.dir_dy = vertical / fbSize.y;
  }

  __global__ void g_localCompositing(FinalCompositingResult *results,
                                     const Fragment *fragments_allRanksMyPixels,
                                     int numPixels,
                                     int numRanks)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numPixels) return;
    
    const Fragment *myFragments = fragments_allRanksMyPixels + tid * numRanks;
    FinalCompositingResult *myResult = results+tid;
    myResult->value = 0.f;
    for (int depth=0;depth<numRanks;depth++)
      myResult->value += myFragments[depth].value;
  }
  

  void localCompositing(FinalCompositingResult *results,
                        const Fragment *fragments,
                        int numPixels,
                        int numRanks)
  {
    int bs = 128;
    int nb = divRoundUp(numPixels,bs);
    g_localCompositing<<<nb,bs>>>(results,fragments,numPixels,numRanks);
  }


  void createModel(std::vector<vec3f> &vertices,
                   std::vector<vec3i> &indices,
                   int thisRank, int numRanks)
  {
    size_t FNV_PRIME = 0x00000100000001b3ull;

    float rectOffset = -1.f;
    float rectSpacing = 2.f/numRanks;
    float rectSize = 1.f / numRanks;
    
    float shiftPerDepth = .8f / numRanks;

    auto addBox = [&](int x, int y, int z)
    {
      float x0 = rectOffset + x * rectSpacing + z * shiftPerDepth;
      float y0 = rectOffset + y * rectSpacing + z * shiftPerDepth;
      float x1 = x0 + rectSize;
      float y1 = y0 + rectSize;

      int i0 = indices.size();
      vertices.push_back(vec3f(x0,y0,z));
      vertices.push_back(vec3f(x0,y1,z));
      vertices.push_back(vec3f(x1,y0,z));
      vertices.push_back(vec3f(x1,y1,z));
      indices.push_back(vec3i(i0)+vec3i(0,1,3));
      indices.push_back(vec3i(i0)+vec3i(0,3,2));
    };
    for (int z=0;z<numRanks;z++)
      for (int y=0;y<numRanks;y++)
        for (int x=0;x<numRanks;x++) {
          size_t hash = 0x12345;
          hash = hash * FNV_PRIME ^ (x+123);
          hash = hash * FNV_PRIME ^ (y+456);
          int owner = (z + hash) % numRanks;
          if (owner == thisRank)
            addBox(x,y,z);
        }
  }

  void setScene(MPI_Comm comm,      
                faceIteration::Context *fit)
  {
    int rank, size;
    MPI_Comm_rank(comm,&rank);
    MPI_Comm_size(comm,&size);
#if 1
    // this is where you'd set your scene geometry ....
#else
    std::vector<vec3i> indices;
    std::vector<vec3f> vertices;
    createModel(vertices,indices,rank,size);
#endif
  }

  int main(int ac, char **av)
  {
    MPI_Comm comm = MPI_COMM_WORLD;
    
    // =============================================================================
    // init GPU - probably need to do some cleverness to figure ouw
    // which GPU you want to use per rank. or rely on
    // CUDA_VISIBLE_DEVICES being set...
    // =============================================================================
    int gpuID = 0;
    hipSetDevice(gpuID);
    hipFree(0);

    // =============================================================================
    // nit MPI - do this after gpu init so mpi can pick up on gpu.
    // =============================================================================
    int required = MPI_THREAD_MULTIPLE;
    int provided = 0;
    MPI_Init_thread(&ac,&av,required,&provided);
    
    // =============================================================================
    // initialize out compositing context
    // =============================================================================
    CompositingContext *comp
      = new CompositingContext(comm,
                               localCompositing);
    Fragment *localFB = comp->resize(fbSize);

    // =============================================================================
    // specify the geometry
    // =============================================================================
    faceIteration::Context *fit
      = faceIteration::Context::init(gpuID,sizeof(UserMeshData),1,
                                     sizeof(PerLaunchData),
                                     devCode_ptx,
                                     "launchOneRay");
    setScene(comm,fit);
    
    // =============================================================================
    // set up a launch, and issue launch to render local frame buffer
    // =============================================================================
    PerLaunchData launchData;
    launchData.localFB = localFB;
    setCamera(launchData);
    fit->launch(fbSize,&launchData);


    // =============================================================================
    // composite the local frame buffers
    // =============================================================================
    FinalCompositingResult *composited
      = comp->run();
    

    // =============================================================================
    // and wind down in reverse order
    // =============================================================================
    delete fit;
    delete comp;
    
    MPI_Finalize();
    return 0;
  }
  
}
